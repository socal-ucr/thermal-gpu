#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// Includes
#include <stdio.h>
#include <string>

// includes, project
#include "include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
#include "include/argparse.hpp"
#include "include/repeat.h"
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 20
#define ITERATIONS 100000000
#define SMID 5
#include "include/ContAcq-IntClk.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void RandomInit(unsigned*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions

__device__ uint get_smid(void) {

     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}


////////////////////////////////////////////////////////////////////////////////
//Funcational benchmarks

__global__ void SM(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=A[i];
    unsigned int I1=A[i];
    unsigned int I2=B[i];
    float Value2=0;
    float Value=0;
    unsigned I3 = 0;
    // exponential function
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
	{
            repeat2048(asm volatile ("sin.approx.ftz.f32 %0, %2;\n\t"
				     "sin.approx.ftz.f32 %1, %3;" :
				     "=f"(Value2),"=f"(Value) : "f" (Value1), "f"(Value2));
                       asm volatile ("add.rz.f32 %0, %1, %2;": "=f"(Value) : "f"(Value1), "f"(Value2));
                       asm volatile ("add.u32 %0, %1, %2;": "=r"(I3) : "r"(I1), "r"(I2));
                      )
        }
    }
   Value=I3;		

    C[i]=Value;
    __syncthreads();
}
__global__ void SFU_EXP(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=A[i];
    float Value2=0;
    float Value3=0;
    float Value=0;
    // exponential function
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
	{
            repeat2048(Value2=expf(Value1);Value3=expf(Value2);Value1=expf(Value3);)
        }
    }
   Value=Value3-Value2;		

    C[i]=Value;
    __syncthreads();
}
__global__ void SFU_LOG(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float Value2=0;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];


    // logarithmic
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++)
        {
	    repeat2048(asm volatile ("lg2.approx.ftz.f32 %0, %2;\n\t"
				     "lg2.approx.ftz.f32 %1, %3;" :
				     "=f"(Value1),"=f"(Value2) : "f" (I1), "f"(I2));)
        }
    }

   Value=Value3-Value2+Value1;		

    C[i]=Value;
    __syncthreads();

}

__global__ void SFU_SIN(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=A[i];
    float Value2=0;
    float Value3=0;
    float Value=0;

    //sinusoidal functions
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {  
	    repeat2048(asm volatile ("sin.approx.ftz.f32 %0, %2;\n\t"
				     "sin.approx.ftz.f32 %1, %3;" :
				     "=f"(Value2),"=f"(Value3) : "f" (Value1), "f"(Value2));)  
        }
    }


   Value=Value3-Value2+Value1;		

    C[i]=Value;
    __syncthreads();

}

__global__ void SFU_SQRT(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float Value2=9999999;
    float Value3=9999999;
    float Value=0;

    //square root
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("sqrt.approx.ftz.f32 %0, %2;\n\t"
				     "sqrt.approx.ftz.f32 %1, %3;" :
				     "=f"(Value),"=f"(Value1) : "f" (Value2), "f"(Value3));)
        }
    }
   Value=Value3-Value2+Value1;		

    C[i]=Value;
    __syncthreads();

}
__global__ void FP_ADD(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float I1=A[i];
    float I2=B[i];

    unsigned int smid = get_smid();

    if (smid == SMID)
    {
        // Excessive Addition access
    	for(unsigned long k=0; k<ITERATIONS;k++)
	{
	    repeat2048(asm volatile ("add.rz.f32 %0, %1, %2;": "=f"(Value1) : "f"(I1), "f"(I2));)
    	}
    }
    __syncthreads();
    C[i]=Value1;
}

__global__ void FP_DIV(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1 = 0.0;
    float I1=A[i];
    float I2=B[i];


    __syncthreads();
   // Excessive Division Operations
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("div.rz.f32 %0, %1, %2;": "=f"(Value1) : "f"(I1), "f"(I2));)
        }
    }
    __syncthreads();
    C[i]= Value1;
}

__global__ void FP_MAD(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value;
    float I1=A[i];
    float I2=B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) {
            repeat2048(asm volatile ("fma.rz.f32 %0, %1, %2, %2;": "=f"(Value) : "f"(I1), "f"(I2));)
        }
    }
    __syncthreads();

    C[i]=Value;
}

__global__ void FP_MUL(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1;
    float I1=A[i];
    float I2=B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++)
        {
	    repeat2048(asm volatile ("mul.rz.f32 %0, %1, %2;": "=f"(Value1) : "f"(I1), "f"(I2));)
        }
    }
    __syncthreads();

    C[i]=Value1;

}

__global__ void INT_ADD(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned I1=(unsigned)A[i];
    unsigned I2=(unsigned)B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("add.u32 %0, %1, %2;": "=r"(Value1) : "r"(I1), "r"(I2));)
        }
    }
    __syncthreads();

    C[i]=(float)Value1;
}
__global__ void INT_DIV(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned I1=(unsigned)A[i];
    unsigned I2=(unsigned)B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("div.u32 %0, %1, %2;": "=r"(Value1) : "r"(I1), "r"(I2));)
        }
    }
    __syncthreads();

    C[i]=(float)Value1;
}
__global__ void INT_LOGIC(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned I1=(unsigned)A[i];
    unsigned I2=(unsigned)B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("and.b32 %0, %1, %2;": "=r"(Value1) : "r"(I1), "r"(I2));)
	    repeat2048(asm volatile ("or.b32 %0, %1, %2;": "=r"(Value1) : "r"(I1), "r"(I2));)
        }
    }
    __syncthreads();

    C[i]=(float)Value1;

}
__global__ void INT_MUL(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned I1=(unsigned)A[i];
    unsigned I2=(unsigned)B[i];

    // Excessive Addition access
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS;k++) 
        {
	    repeat2048(asm volatile ("mul.lo.u32 %0, %1, %2;": "=r"(Value1) : "r"(I1), "r"(I2));)
        }
    }
    __syncthreads();

    C[i]=(float)Value1;
}

///////////////////// CACHE and MEMORY BENCHMARKS/////////////////////////////////
__global__ void L1(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned long k=0;
    // Excessive Addition access
    unsigned int smid = get_smid();
    int temp = 0;
    if(smid == SMID)
    {
	// Fill the L1 cache, Miss on first LD, Hit on subsequent LDs
	for(k=0; k<ITERATIONS; ++k){
            repeat2048(asm volatile ("ld.global.u32 %0, [%1];" : "=r"(temp): "l" (A+tid));)
	}

	C[tid]=temp;
    }
    __syncthreads();
}
__global__ void L1_ALL(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned k=0;
    // Excessive Addition access
    unsigned int smid = get_smid();
    int temp = 0;
    // Fill the L1 cache, Miss on first LD, Hit on subsequent LDs
    for(k=0; k<ITERATIONS; ++k){
        repeat2048(asm volatile ("ld.global.u32 %0, [%1];" : "=r"(temp): "l" (A+tid));)
    }

    C[tid]=temp;
    __syncthreads();
}

__global__ void L2(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned long k=0;
    // Excessive Addition access
    unsigned int smid = get_smid();
    int temp = 0;
    if(smid == SMID)
    {
	// Fill the L1 cache, Miss on first LD, Hit on subsequent LDs
	for(k=0; k<ITERATIONS; ++k){
            repeat2048(asm volatile ("ld.cg.u32 %0, [%1];" : "=r"(temp): "l" (A+tid));)
	}

	C[tid]=temp;
    }
    __syncthreads();
}

__global__ void L2_ALL(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned k=0;
    int temp = 0;
    for(k=0; k<ITERATIONS; ++k){
        repeat2048(asm volatile ("ld.cg.u32 %0, [%1];" : "=r"(temp): "l" (A+tid));)
    }

    C[tid]=temp;
    __syncthreads();
}

__global__ void I_CACHE(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    int temp = 0;
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(unsigned long k=0; k<ITERATIONS; ++k){
LABEL:
            goto LABEL;
        }
    }

    C[tid]=temp;
    __syncthreads();
}

__global__ void REG_FILE(float* A, float* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned long k = 0;
    unsigned long temp = 123456789;
    unsigned long temp1 = 0;
    unsigned int smid = get_smid();
    if(smid == SMID)
    {
        for(k=0; k<ITERATIONS; ++k){
            repeat2048(asm volatile ("mov.u64 %0, %1;" : "=l"(temp1): "l" (temp));)
        }
    }
    k = temp1;
    C[tid]=temp;
    __syncthreads();
}
__global__ void SHD_MEM(float* A, float* C, int N){
    //int size = (LINE_SIZE*ASSOC*SETS)/sizeof(int);
    //unsigned j=0, k=0;
    unsigned long k=0;
    unsigned int smid = get_smid();

    __shared__ unsigned long long sdata[THREADS_PER_BLOCK];

    __shared__ void **tmp_ptr;

    __shared__ void *arr[THREADS_PER_BLOCK];
    int i =0; 
    if (threadIdx.x == 0) {
        for (i=0; i < THREADS_PER_BLOCK; i++) {
            arr[i] = (void *)&sdata[i];
        }
        for (i=0; i < (THREADS_PER_BLOCK - 1); i++) {
            sdata[i] = (unsigned long long)arr[i+1];
        }
        sdata[THREADS_PER_BLOCK - 1] = (unsigned long long) arr[0];
    }

    __syncthreads();

    tmp_ptr = (void **)(&(arr[(threadIdx.x + 1)%THREADS_PER_BLOCK]));
    if(smid == SMID)
    {
        for(k=0; k<ITERATIONS; ++k){
            repeat2048(tmp_ptr = (void**)(*tmp_ptr);)
        }
    }
    __syncthreads();
}

int main(int argc, const char** argv)
{
    ArgumentParser parser;
    parser.addArgument("-t","--test",1,false);

    parser.parse(argc, argv);
    std::string test_name = parser.retrieve<std::string>("test");
    int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
    size_t size = N * sizeof(float);
    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) CleanupResources();
    h_B = (float*)malloc(size);
    if (h_B == 0) CleanupResources();
    h_C = (float*)malloc(size);
    if (h_C == 0) CleanupResources();

    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
    checkCudaErrors( hipMalloc((void**)&d_A, size) );
    checkCudaErrors( hipMalloc((void**)&d_B, size) );
    checkCudaErrors( hipMalloc((void**)&d_C, size) );

    // Copy vectors from host memory to device memory
    checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

    dim3 dimGrid(NUM_OF_BLOCKS,1,1);
    dim3 dimBlock(THREADS_PER_BLOCK,1,1);

    printf("Microbenchmarks-%s\n",test_name.c_str());
    if(test_name.compare("SM") == 0)
        SM<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("SFU_EXP") == 0)
        SFU_EXP<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("SFU_LOG") == 0)
        SFU_LOG<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("SFU_SIN") == 0)
        SFU_SIN<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("SFU_SQRT") == 0)
        SFU_SQRT<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("FP_ADD") == 0)
        FP_ADD<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("FP_DIV") == 0)
        FP_DIV<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("FP_MAD") == 0)
        FP_MAD<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("FP_MUL") == 0)
        FP_MUL<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("INT_ADD") == 0)
        INT_ADD<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("INT_DIV") == 0)
        INT_DIV<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("INT_LOGIC") == 0)
        INT_LOGIC<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("INT_MUL") == 0)
        INT_MUL<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    else if(test_name.compare("L1") == 0)
        L1<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("L1_ALL") == 0)
        L1_ALL<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("L2") == 0)
        L2<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("L2_ALL") == 0)
        L2_ALL<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("I_CACHE") == 0)
        I_CACHE<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("REG_FILE") == 0)
        REG_FILE<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else if(test_name.compare("SHD_MEM") == 0)
        SHD_MEM<<<dimGrid,dimBlock>>>(d_A, d_C, N);
    else
    {
        printf("INVALID TEST\n");
        exit(1);
    }

    getLastCudaError("kernel launch failure");
    hipDeviceSynchronize();
    printf("after\n");

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
     
    CleanupResources();

    return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i){ 
	data[i] = rand() / RAND_MAX;
  }
}
